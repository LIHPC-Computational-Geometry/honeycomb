#include "hip/hip_runtime.h"
#include "../include/honeycomb.h"
#include <stdint.h>

// BLOCK: (2, 2, 24)
// GRID:  (ceil(n_x/2), ceil(n_y/2), n_z)
extern "C" __global__ void
generate_hex_grid_betaf(DartIdType* out, size_t n_x, size_t n_y, size_t n_z, size_t n_out) {
  int offset_x           = 24;
  int offset_y           = offset_x * n_x;
  int offset_z           = offset_y * n_y;
  int const OFFSET_X     = 24;
  int const BETAS[24][4] = {
    // 1st face
    {3, 1, 4, 20},
    {-1, 1, 7, 22},
    {-1, 1, 10, 20},
    {-1, -3, 13, 18},
    // 2nd face
    {3, 1, -4, 8},
    {-1, 1, 14, 10},
    {-1, 1, 14, 8},
    {-1, -3, 2, 6},
    // 3rd face
    {3, 1, -7, 8},
    {-1, 1, -2, 10},
    {-1, 1, 13, 8},
    {-1, -3, 2, 6},
    // 4th face
    {3, 1, -10, -8},
    {-1, 1, -2, -6},
    {-1, 1, 8, -8},
    {-1, -3, 2, -10},
    // 5th face
    {3, 1, -13, -8},
    {-1, 1, -2, -6},
    {-1, 1, 3, -8},
    {-1, -3, -14, -10},
    // 6th face
    {3, 1, -14, -20},
    {-1, 1, -3, -18},
    {-1, 1, -8, -20},
    {-1, -3, -13, -22},
  };
  int offsets[24] = {-offset_y, -offset_y, -offset_y, -offset_y, -offset_z, -offset_z,
                     -offset_z, -offset_z, +offset_x, +offset_x, +offset_x, +offset_x,
                     +offset_z, +offset_z, +offset_z, +offset_z, -offset_x, -offset_x,
                     -offset_x, -offset_x, +offset_y, +offset_y, +offset_y, +offset_y};
  // cell coordinates in the generated grid
  uint64_t ix = threadIdx.x + blockIdx.x * blockDim.x;
  uint64_t iy = threadIdx.y + blockIdx.y * blockDim.y;
  uint64_t iz = blockIdx.z;
  // dart of the thread
  uint64_t dart = 1 + offset_x * ix + offset_y * iy + offset_z * iz + threadIdx.z;
  // boundary conditions
  int conds[24] = {
    iy == 0,       iy == 0,       iy == 0,       iy == 0,       iz == 0,       iz == 0,
    iz == 0,       iz == 0,       ix == n_x - 1, ix == n_x - 1, ix == n_x - 1, ix == n_x - 1,
    iz == n_z - 1, iz == n_z - 1, iz == n_z - 1, iz == n_z - 1, ix == 0,       ix == 0,
    ix == 0,       ix == 0,       iy == n_y - 1, iy == n_y - 1, iy == n_y - 1, iy == n_y - 1,
  };
  // beta images
  if (dart * 4 + 3 < n_out) {
    out[dart * 4]     = dart + BETAS[threadIdx.z][0];
    out[dart * 4 + 1] = dart + BETAS[threadIdx.z][1];
    out[dart * 4 + 2] = dart + BETAS[threadIdx.z][2];
    out[dart * 4 + 3] =
      conds[threadIdx.z] ? 0 : dart + BETAS[threadIdx.z][3] + offsets[threadIdx.z];
  }
}

// BLOCK: (2, 2, 24)
// GRID:  (ceil(n_x/2), ceil(n_y/2), n_z)
extern "C" __global__ void generate_hex_grid_vertices(
  CuVertex3* out, float lc_x, float lc_y, float lc_z, size_t n_x, size_t n_y, size_t n_out
) {
  int const OFFSETS[24][3] = {
    {0, 0, 0}, {1, 0, 0}, {1, 0, 1}, {0, 0, 1}, {1, 0, 0}, {0, 0, 0}, {0, 1, 0}, {1, 1, 0},
    {1, 0, 1}, {1, 0, 0}, {1, 1, 0}, {1, 1, 1}, {0, 0, 1}, {1, 0, 1}, {1, 1, 1}, {0, 1, 1},
    {0, 0, 0}, {0, 0, 1}, {0, 1, 1}, {0, 1, 0}, {1, 1, 0}, {0, 1, 0}, {0, 1, 1}, {1, 1, 1},
  };
  int offset_x = 24;
  int offset_y = offset_x * n_x;
  int offset_z = offset_y * n_y;

  // cell coordinates in the generated grid
  uint64_t ix = threadIdx.x + blockIdx.x * blockDim.x;
  uint64_t iy = threadIdx.y + blockIdx.y * blockDim.y;
  uint64_t iz = blockIdx.z;
  // dart of the thread
  uint64_t dart = 1 + offset_x * ix + offset_y * iy + offset_z * iz + threadIdx.z;
  // compute the vertex associated to every single dart;
  // we'll filter useful values when building on the host
  if (dart < n_out) {
    out[dart] = {
      lc_x * (ix + OFFSETS[threadIdx.z][0]),
      lc_y * (iy + OFFSETS[threadIdx.z][1]),
      lc_z * (iz + OFFSETS[threadIdx.z][2])
    };
  }
}
