#include "hip/hip_runtime.h"
#include "../include/honeycomb.h"
#include <stdint.h>

// BLOCK: (4, 4, 4)
// GRID:  (ceil(n_x/2), ceil(n_y/2), 1)
extern "C" __global__ void generate_2d_grid_betaf(DartIdType* out, size_t n_x, size_t n_y, size_t n_out) {
    const int BETAS[4][3] = { {3, 1, 2}, {-1, 1, 6}, {-1, 1, -2}, {-1, -3, -6} };
    const int NX[4]       = { -4, 0, 4, 0 };
    // cell coordinates in the generated grid
    uint64_t ix = threadIdx.x + blockIdx.x * blockDim.x;
    uint64_t iy = threadIdx.y + blockIdx.y * blockDim.y;
    // dart of the thread
    uint64_t dart = 1 + 4 * ix + 4 * n_x * iy + threadIdx.z;
    // boundary conditions
    int conds[4] = { iy == 0, ix == n_x - 1, iy == n_y - 1, ix == 0 };
    // beta images
    if (dart*3 + 2 < n_out) {
        out[dart*3]   = dart + BETAS[threadIdx.z][0];
        out[dart*3+1] = dart + BETAS[threadIdx.z][1];
        out[dart*3+2] = conds[threadIdx.z] ? 0 : dart + BETAS[threadIdx.z][2] + NX[threadIdx.z] * n_x;
    }
}

// BLOCK: (4, 4, 4)
// GRID:  (ceil(n_x/2), ceil(n_y/2), 1)
extern "C" __global__ void generate_2d_grid_vertices(
    CuVertex2* out,
    float lc_x,
    float lc_y,
    size_t n_x,
    size_t n_out
) {
    const int OFFSETS[4][2] = { {0, 1}, {0, 0}, {1, 0}, {1, 1} };
    // cell coordinates in the generated grid
    uint64_t ix = threadIdx.x + blockIdx.x * blockDim.x;
    uint64_t iy = threadIdx.y + blockIdx.y * blockDim.y;
    // dart of the thread
    uint64_t dart = 1 + 4 * ix + 4 * n_x * iy + threadIdx.z;
    // compute the vertex associated to every single dart;
    // we'll filter useful values when building on the host
    if (dart < n_out) {
        out[dart] = {
            lc_x * (ix + OFFSETS[threadIdx.z][0]),
            lc_y * (iy + OFFSETS[threadIdx.z][1])
        };
    }
}
